
#include <hip/hip_runtime.h>
#include <stdio.h>

// We move the body of collide into a separate device function collide_body to allow us
// to pass collide_body to various differentiation methods. This is necessary as differentiation
// can only be done on device, not global kernel functions.
__device__
void collide_body(float* src, float* dst) {
    size_t idx = threadIdx.x;
    if (idx < 100) {
        dst[idx] += src[idx] * src[idx] - 3 * src[idx];
    }
}

// GPU Kernel
__global__
void collide(float* src, float* dst) {
    collide_body(src, dst);
}

// Wrapper CPU function which calls kernel
__attribute__((noinline))
void kern(float* src, float* dst) {
    collide<<<1, 100>>>(src, dst);
}

// // Main CPU code that calls wrapper function
// void iter(int nTimeSteps, float* src, float* dst) {
//     for (unsigned int i=0; i<nTimeSteps/2; i++) {
//         kern(src, dst);
//         kern(dst, src);
//     }
// }
// 
// template <typename... Args>
// void __enzyme_autodiff(Args...);
// 
// void grad_iter(int nTimeSteps, float* src, float* dsrc, float* dst, float* ddst) {
//     __enzyme_autodiff(iter, nTimeSteps, src, dsrc, dst, ddst);
// }

// A function similar to __enzyme_autodiff, except it only calls the augmented forward pass, returning
// a tape structure to hold any values that may be overwritten and needed for the reverse.
template <typename... Args>
__device__ void* __enzyme_augmentfwd(Args...);

// A function similar to __enzyme_autodiff, except it only calls the revese pass, taking in the tape
// as its last argument.
template <typename... Args>
__device__ void __enzyme_reverse(Args...);

// A wrapper GPU kernel for calling the forward pass of collide. The wrapper code stores
// the tape generated by Enzyme into a unique location per thread
__global__ void aug_collide(float* src, float* dsrc, float* dst, float* ddst, void** tape)
{
    size_t idx = threadIdx.x;
    tape[idx] = __enzyme_augmentfwd((void*)collide_body, src, dsrc, dst, ddst);
}

// A wrapper GPU kernel for calling the reverse pass of collide. The wrapper code retrieves
// the corresponding tape per thread being executed.
__global__ void rev_collide( float* src, float* dsrc, float* dst, float* ddst, void** tape)
{
    size_t idx = threadIdx.x;
    __enzyme_reverse((void*)collide_body, src, dsrc, dst, ddst, tape[idx]);
}

// The augmented forward pass of the CPU kern call, allocating and returning
// tape memory  needed to compute the reverse pass. This calls a augmented collide
// GPU kernel, passing in a unique 8-byte location to store the tape.
void* aug_kern(float* src, float* dsrc, float* dst, float* ddst) {
    void** tape;
    hipMalloc(&tape, sizeof(void*) * /*total number of threads*/100);
    aug_collide<<<1, 100>>>(src, dsrc, dst, ddst, tape);
    return (void*)tape;
}

// The reverse pass of the CPU kern call, using tape memory passed as the
// last argument. This calls a reverse collide GPU kernel.
void rev_kern(float* src, float* dsrc, float* dst, float* ddst, void* tape) {
    rev_collide<<<1, 100>>>(src, dsrc, dst, ddst, (void**)tape);
    hipFree(tape);
}

// Here we register the custom forward pass aug_kern and reverse pass rev_kern
void* __enzyme_register_gradient_kern[3] = { (void*)kern, (void*)aug_kern, (void*)rev_kern };

int main() {

    double *x, *d_x, *y, *d_y; // device pointers

    hipMalloc(&x, sizeof(*x));
    hipMalloc(&d_x, sizeof(*d_x));
    hipMalloc(&y, sizeof(*y));
    hipMalloc(&d_y, sizeof(*d_y));

    double host_x = 1.4;
    double host_d_x = 0.0;
    double host_y;
    double host_d_y = 1.0;

    hipMemcpy(x,   &host_x,   sizeof(*x),   hipMemcpyHostToDevice);
    hipMemcpy(d_x, &host_d_x, sizeof(*d_x), hipMemcpyHostToDevice);
    hipMemcpy(y,   &host_y,   sizeof(*y),   hipMemcpyHostToDevice);
    hipMemcpy(d_y, &host_d_y, sizeof(*d_y), hipMemcpyHostToDevice);

    // ToDo

    hipDeviceSynchronize(); // synchroniz

    hipMemcpy(&host_x,   x,   sizeof(*x),   hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_x, d_x, sizeof(*d_x), hipMemcpyDeviceToHost);
    hipMemcpy(&host_y,   y,   sizeof(*y),   hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_y, d_y, sizeof(*d_y), hipMemcpyDeviceToHost);

    printf("%f %f\n", host_x,   host_y);
    printf("%f %f\n", host_d_x, host_d_y);
}
