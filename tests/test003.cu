// This is the CUDA version of test002.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>

void __device__ foo_impl(double *x_in, double *x_out) {
    x_out[0] = x_in[0] * x_in[0];
}

typedef void (*f_ptr)(double *, double *);

extern void __device__ __enzyme_autodiff(f_ptr,
    int, double *, double *,
    int, double *, double *
);

void __global__ foo(double *x_in, double *x_out) {
    foo_impl(x_in, x_out);
}

int __device__ enzyme_dup;
int __device__ enzyme_out;
int __device__ enzyme_const;

void __global__ foo_grad(double *x, double *d_x, double *y, double *d_y) {

    __enzyme_autodiff(foo_impl,
        enzyme_dup, x, d_x,
        enzyme_dup, y, d_y);
}

int main() {

    double *x, *d_x, *y, *d_y; // device pointers

    hipMalloc(&x, sizeof(*x));
    hipMalloc(&d_x, sizeof(*d_x));
    hipMalloc(&y, sizeof(*y));
    hipMalloc(&d_y, sizeof(*d_y));

    double host_x = 1.4;
    double host_d_x = 0.0;
    double host_y;
    double host_d_y = 1.0;

    hipMemcpy(x,   &host_x,   sizeof(*x),   hipMemcpyHostToDevice);
    hipMemcpy(d_x, &host_d_x, sizeof(*d_x), hipMemcpyHostToDevice);
    hipMemcpy(y,   &host_y,   sizeof(*y),   hipMemcpyHostToDevice);
    hipMemcpy(d_y, &host_d_y, sizeof(*d_y), hipMemcpyHostToDevice);

    // foo<<<1,1>>>(x, y); fwd-pass only
    foo_grad<<<1,1>>>(x, d_x, y, d_y); // fwd and bkwd pass

    hipDeviceSynchronize(); // synchroniz

    hipMemcpy(&host_x,   x,   sizeof(*x),   hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_x, d_x, sizeof(*d_x), hipMemcpyDeviceToHost);
    hipMemcpy(&host_y,   y,   sizeof(*y),   hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_y, d_y, sizeof(*d_y), hipMemcpyDeviceToHost);

    printf("%f %f\n", host_x,   host_y);
    printf("%f %f\n", host_d_x, host_d_y);
}

/*

clang test003.cu -Xclang -load -Xclang /home/linuxbrew/.linuxbrew/Cellar/enzyme/0.0.19/lib/ClangEnzyme-12.so -O2 -fno-vectorize -fno-unroll-loops -fPIC --cuda-gpu-arch=sm_70 -lcudart -L/usr/local/cuda-11.3/lib64; ./a.out

*/
